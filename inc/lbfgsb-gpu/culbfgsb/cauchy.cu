#include "hip/hip_runtime.h"
/**
 * \copyright 2012 Yun Fei
 * in collaboration with G. Rong, W. Wang and B. Wang
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */

#ifdef __INTELLISENSE__
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#endif

#include "culbfgsb/lbfgsbcuda.h"

namespace lbfgsbcuda {
namespace cuda {
namespace cauchy {

template <int bx, typename real>
__global__ void kernel0(int n, const real* g, const int* nbd, real* t,
                        const real* x, const real* u, const real* l,
                        int* iwhere, const real machinemaximum) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  const int tid = threadIdx.x;
  volatile __shared__ real sdata[bx];

  real mySum;

  if (i < n) {
    int iwi = iwhere[i];
    if (iwi != 3 && iwi != -1) {
      real neggi = -g[i];
      int nbdi = nbd[i];

      real tl = 0;
      real tu = 0;

      if (nbdi <= 2) {
        tl = x[i] - l[i];
      }
      if (nbdi >= 2) {
        tu = u[i] - x[i];
      }

      if (nbdi <= 2 && tl <= 0 && neggi <= 0) {
        iwi = 1;
      } else if (nbdi >= 2 && tu <= 0 && neggi >= 0) {
        iwi = 2;
      } else if (neggi == 0) {
        iwi = -3;
      } else {
        iwi = 0;
      }

      iwhere[i] = iwi;

      if ((iwi != 0 && iwi != -1) || neggi == 0) {
        mySum = machinemaximum;
      } else {
        if (nbdi <= 2 && nbdi != 0 && neggi < 0) {
          mySum = tl / (-neggi);
        } else if (nbdi >= 2 && neggi > 0) {
          mySum = tu / neggi;
        } else {
          mySum = machinemaximum;
        }
      }
    } else {
      mySum = machinemaximum;
    }
  } else {
    mySum = machinemaximum;
  }

  sdata[tid] = mySum;
  __syncthreads();

  if (bx > 512) {
    if (tid < 512) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (bx > 256) {
    if (tid < 256) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (bx > 128) {
    if (tid < 128) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (bx > 64) {
    if (tid < 64) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < min(bx / 2, 32)) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile real* smem = sdata + tid;
    if (bx > 32) {
      *smem = mySum = minr(mySum, smem[32]);
    }
    if (bx > 16) {
      *smem = mySum = minr(mySum, smem[16]);
    }
    if (bx > 8) {
      *smem = mySum = minr(mySum, smem[8]);
    }
    if (bx > 4) {
      *smem = mySum = minr(mySum, smem[4]);
    }
    if (bx > 2) {
      *smem = mySum = minr(mySum, smem[2]);
    }
    if (bx > 1) {
      *smem = mySum = minr(mySum, smem[1]);
    }
  }

  if (tid == 0) t[blockIdx.x] = mySum;
}

template <int bx, typename real>
__global__ void kernel01(const int n, const real* buf_in, real* buf_out,
                         const real machinemaximum) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int tid = threadIdx.x;

  volatile __shared__ real sdata[bx];

  real mySum;

  if (i < n)
    mySum = buf_in[i];
  else
    mySum = machinemaximum;

  sdata[tid] = mySum;
  __syncthreads();
  if (bx > 512) {
    if (tid < 512) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (bx > 256) {
    if (tid < 256) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (bx > 128) {
    if (tid < 128) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (bx > 64) {
    if (tid < 64) {
      sdata[tid] = mySum = minr(mySum, sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < min(bx / 2, 32)) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile real* smem = sdata + tid;
    if (bx > 32) {
      *smem = mySum = minr(mySum, smem[32]);
    }
    if (bx > 16) {
      *smem = mySum = minr(mySum, smem[16]);
    }
    if (bx > 8) {
      *smem = mySum = minr(mySum, smem[8]);
    }
    if (bx > 4) {
      *smem = mySum = minr(mySum, smem[4]);
    }
    if (bx > 2) {
      *smem = mySum = minr(mySum, smem[2]);
    }
    if (bx > 1) {
      *smem = mySum = minr(mySum, smem[1]);
    }
  }

  if (tid == 0) {
    buf_out[blockIdx.x] = mySum;
  }
}

template <int bx, typename real>
__global__ void kernel1(const int n, const real* g, real* buf_s_r,
                        const int* iwhere) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  const int tid = threadIdx.x;
  volatile __shared__ real sdata[bx];

  real mySum;

  if (i >= n) {
    mySum = 0;
  } else {
    int iwi = iwhere[i];
    if (iwi != 0 && iwi != -1) {
      mySum = 0;
    } else {
      real neggi = g[i];
      mySum = -neggi * neggi;
    }
  }

  sdata[tid] = mySum;
  __syncthreads();

  if (bx > 512) {
    if (tid < 512) {
      sdata[tid] = mySum = (mySum + sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (bx > 256) {
    if (tid < 256) {
      sdata[tid] = mySum = (mySum + sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (bx > 128) {
    if (tid < 128) {
      sdata[tid] = mySum = (mySum + sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (bx > 64) {
    if (tid < 64) {
      sdata[tid] = mySum = (mySum + sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < min(bx / 2, 32)) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile real* smem = sdata + tid;
    if (bx > 32) {
      *smem = mySum = mySum + smem[32];
    }
    if (bx > 16) {
      *smem = mySum = mySum + smem[16];
    }
    if (bx > 8) {
      *smem = mySum = mySum + smem[8];
    }
    if (bx > 4) {
      *smem = mySum = mySum + smem[4];
    }
    if (bx > 2) {
      *smem = mySum = mySum + smem[2];
    }
    if (bx > 1) {
      *smem = mySum = mySum + smem[1];
    }
  }

  if (tid == 0) buf_s_r[blockIdx.x] = mySum;
}

template <int bx, typename real>
__global__ void kernel20(const int n, const int head, const int m,
                         const int col, const int iPitch, const int oPitch,
                         const real* g, real* buf_array_p, const real* wy,
                         const real* ws, const int* iwhere) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y;
  const int tid = threadIdx.x;

  volatile __shared__ real sdata[bx];

  real mySum;

  if (i < n) {
    int iwi = iwhere[i];
    if (iwi != 0 && iwi != -1) {
      mySum = 0;
    } else {
      real neggi = -g[i];

      real p0;
      if (j < col) {
        int pointr = Modular((head + j), m);
        p0 = wy[i * iPitch + pointr];
      } else {
        int pointr = Modular((head + j - col), m);
        p0 = ws[i * iPitch + pointr];
      }

      mySum = p0 * neggi;
    }
  } else {
    mySum = 0;
  }

  sdata[tid] = mySum;
  __syncthreads();
  if (bx > 512) {
    if (tid < 512) {
      sdata[tid] = mySum = (mySum + sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (bx > 256) {
    if (tid < 256) {
      sdata[tid] = mySum = (mySum + sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (bx > 128) {
    if (tid < 128) {
      sdata[tid] = mySum = (mySum + sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (bx > 64) {
    if (tid < 64) {
      sdata[tid] = mySum = (mySum + sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < min(bx / 2, 32)) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile real* smem = sdata + tid;
    if (bx > 32) {
      *smem = mySum = mySum + smem[32];
    }
    if (bx > 16) {
      *smem = mySum = mySum + smem[16];
    }
    if (bx > 8) {
      *smem = mySum = mySum + smem[8];
    }
    if (bx > 4) {
      *smem = mySum = mySum + smem[4];
    }
    if (bx > 2) {
      *smem = mySum = mySum + smem[2];
    }
    if (bx > 1) {
      *smem = mySum = mySum + smem[1];
    }
  }

  if (tid == 0) buf_array_p[j * oPitch + blockIdx.x] = mySum;
}

template <int bx, typename real>
__global__ void kernel21(const int n, const int iPitch, const int oPitch,
                         const real* buf_in, real* buf_out) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y;
  const int tid = threadIdx.x;

  volatile __shared__ real sdata[bx];

  real mySum;

  if (i < n)
    mySum = buf_in[j * iPitch + i];
  else
    mySum = 0;

  sdata[tid] = mySum;
  __syncthreads();
  if (bx > 512) {
    if (tid < 512) {
      sdata[tid] = mySum = (mySum + sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (bx > 256) {
    if (tid < 256) {
      sdata[tid] = mySum = (mySum + sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (bx > 128) {
    if (tid < 128) {
      sdata[tid] = mySum = (mySum + sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (bx > 64) {
    if (tid < 64) {
      sdata[tid] = mySum = (mySum + sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < min(bx / 2, 32)) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile real* smem = sdata + tid;
    if (bx > 32) {
      *smem = mySum = mySum + smem[32];
    }
    if (bx > 16) {
      *smem = mySum = mySum + smem[16];
    }
    if (bx > 8) {
      *smem = mySum = mySum + smem[8];
    }
    if (bx > 4) {
      *smem = mySum = mySum + smem[4];
    }
    if (bx > 2) {
      *smem = mySum = mySum + smem[2];
    }
    if (bx > 1) {
      *smem = mySum = mySum + smem[1];
    }
  }

  if (tid == 0) {
    buf_out[j * oPitch + blockIdx.x] = mySum;
  }
}

template <typename real>
__global__ void kernel22(const int n, real* p, const real theta) {
  const int i = threadIdx.x;

  if (i >= n) return;

  p[i] *= theta;
}

template <typename real>
__global__ void kernel4(const int col2, const real* p, real* c,
                        const real dtm) {
  const int i = threadIdx.x;

  if (i >= col2) return;

  c[i] = p[i] * dtm;
}

template <typename real>
__global__ void kernel3(const int n, const real* x, const real* g, real* xcp,
                        real* xcpb, const real dtm, const int* iwhere) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= n) return;

  real inc;
  int iwi = iwhere[i];
  if (iwi != 0 && iwi != -1) {
    inc = 0;
  } else {
    real neggi = -g[i];
    inc = neggi * dtm;
  }
  real res = x[i] + inc;
  xcp[i] = res;
  xcpb[i] = res;
}

template <typename real>
void prog0(const int& n, const real* x, const real* l, const real* u,
           const int* nbd, const real* g, real* t, real* xcp, real* xcpb,
           const int& m, const real* wy, const real* ws, const real* sy,
           const int iPitch, real* wt, const real& theta, const int& col,
           const int& head, real* p, real* c, real* v, int& nint,
           const real& sbgnrm, real* buf_s_r, real* buf_array_p, int* iwhere,
           const int& iPitch_normal, const real& machinemaximum,
           hipblasHandle_t cublas_handle, const hipStream_t* streamPool) {
  debugSync();

  if (sbgnrm <= 0) {
    hipMemcpyAsync(xcp, x, n * sizeof(real), hipMemcpyDeviceToDevice);
    return;
  }

  if (col > 0) hipMemsetAsync(p, 0, col * 2 * sizeof(real));

  real* vec_h;
  real* vec_d;

  cutilSafeCall(hipHostAlloc(&vec_h, 3 * sizeof(real), hipHostMallocMapped));
  cutilSafeCall(hipHostGetDevicePointer(&vec_d, vec_h, 0));

  real* bkmin_d = vec_d;
  real* f1_d = vec_d + 1;

  real* bkmin_h = vec_h;
  real* f1_h = vec_h + 1;
  real* fd_h = vec_h + 2;

  int nblock0 = n;
  int mi = log2Up(nblock0);
  int nblock1 = iDivUp2(nblock0, mi);

  real* output0 = (nblock1 == 1) ? bkmin_d : t;
  real* output1 = (nblock1 == 1) ? f1_d : buf_s_r;
  real* output2 = (nblock1 == 1) ? p : buf_array_p;

  dynamicCall(kernel0, mi, real, nblock1, 1, streamPool[0],
              (nblock0, g, nbd, output0, x, u, l, iwhere, machinemaximum));

  dynamicCall(kernel1, mi, real, nblock1, 1, streamPool[0],
              (nblock0, g, output1, iwhere));

  int op20 = (nblock1 == 1) ? 1 : iPitch_normal;

  if (col > 0) {
    dynamicCall(
        kernel20, mi, real, nblock1, col * 2, streamPool[0],
        (nblock0, head, m, col, iPitch, op20, g, output2, wy, ws, iwhere));
  }
  nblock0 = nblock1;

  while (nblock0 > 1) {
    nblock1 = iDivUp2(nblock0, mi);

    real* input0 = output0;
    real* input1 = output1;
    real* input2 = output2;

    output0 = (nblock1 == 1) ? bkmin_d : (output0 + nblock0);
    output1 = (nblock1 == 1) ? f1_d : (output1 + nblock0);
    output2 = (nblock1 == 1) ? p : (output2 + nblock0);

    dynamicCall(kernel01, mi, real, nblock1, 1, streamPool[0],
                (nblock0, input0, output0, machinemaximum));

    dynamicCall(kernel21, mi, real, nblock1, 1, streamPool[1],
                (nblock0, 1, 1, input1, output1));

    int op20 = (nblock1 == 1) ? 1 : iPitch_normal;
    if (col > 0) {
      dynamicCall(kernel21, mi, real, nblock1, col * 2, streamPool[2],
                  (nblock0, iPitch_normal, op20, input2, output2));
    }

    nblock0 = nblock1;
  }

  if (col > 0 && theta != 1) {
    debugSync();
    kernel22<real><<<dim3(1), dim3(col), 0, streamPool[2]>>>(col, p + col, theta);

    debugSync();
  }

  *fd_h = 0;

  if (col > 0) {
    bmv::prog0<real>(sy, col, iPitch, p, v, streamPool[2]);

    debugSync();
    bmv::prog1<real>(wt, col, iPitch, p, v, cublas_handle, streamPool[2]);

    debugSync();
    bmv::prog2<real>(sy, wt, col, iPitch, p, v, streamPool[2]);

    debugSync();
    hipblasSetStream(cublas_handle, streamPool[2]);

    cublasRdot<real>(cublas_handle, col * 2, v, 1, p, 1, fd_h);

    hipblasSetStream(cublas_handle, NULL);
  }

  cutilSafeCall(hipDeviceSynchronize());

  real f2 = -theta * *f1_h - *fd_h;
  real dt = -*f1_h / f2;

  real dtm = std::min(*bkmin_h, dt);
  dtm = std::max(static_cast<real>(0.0), dtm);

  kernel3<real><<<dim3(iDivUp(n, 512)), dim3(512), 0, streamPool[0]>>>(
      n, x, g, xcp, xcpb, dtm, iwhere);

  if (col > 0) {
    kernel4<real>
        <<<dim3(1), dim3(col * 2), 0, streamPool[1]>>>(col * 2, p, c, dtm);
  }
}

#define INST_HELPER(real)                                                     \
  template void prog0<real>(                                                  \
      const int&, const real*, const real*, const real*, const int*,          \
      const real*, real*, real*, real*, const int&, const real*, const real*, \
      const real*, const int, real*, const real&, const int&, const int&,     \
      real*, real*, real*, int&, const real&, real*, real*, int*, const int&, \
      const real&, hipblasHandle_t, const hipStream_t*);

INST_HELPER(double);
INST_HELPER(float);
};  // namespace cauchy
};  // namespace cuda
};  // namespace lbfgsbcuda