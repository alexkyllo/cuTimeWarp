#include "hip/hip_runtime.h"
/**
 * \copyright 2012 Yun Fei
 * in collaboration with G. Rong, W. Wang and B. Wang
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */

#include "lbfgsbcuda.h"

namespace lbfgsbcuda {
namespace cuda {
namespace matupd {

template <typename real>
__global__ void kernel0(int n, real* wy, const real* r, const int iPitch) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;

  wy[i * iPitch] = r[i];
}

template <typename real>
__global__ void kernel1(real* sy, const int iPitch_i, const int iPitch_j,
                        const int col) {
  const int i = threadIdx.x;
  const int j = threadIdx.y;

  __shared__ real sdata[8][8];

  sdata[j][i] = sy[j * iPitch_i + i * iPitch_j];

  if (i >= col - 1 || j >= col - 1 || i > j) return;

  __syncthreads();

  sy[j * iPitch_i + i * iPitch_j] = sdata[j + 1][i + 1];
}

template <int bx, typename real>
__global__ void kernel20(const int n, const int head, const int m,
                         const int col, const int iPitch, const int oPitch,
                         const real* d, real* buf_array_p, const real* wy) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y;
  const int tid = threadIdx.x;

  volatile __shared__ real sdata[bx];

  real mySum;

  int pointr = Modular((head + j), m);
  if (i < n) {
    mySum = d[i] * wy[i * iPitch + pointr];
  } else {
    mySum = 0;
  }

  sdata[tid] = mySum;
  __syncthreads();
  if (bx > 512) {
    if (tid < 512) {
      sdata[tid] = mySum = (mySum + sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (bx > 256) {
    if (tid < 256) {
      sdata[tid] = mySum = (mySum + sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (bx > 128) {
    if (tid < 128) {
      sdata[tid] = mySum = (mySum + sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (bx > 64) {
    if (tid < 64) {
      sdata[tid] = mySum = (mySum + sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < min(bx / 2, 32)) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile real* smem = sdata + tid;
    if (bx > 32) {
      *smem = mySum = mySum + smem[32];
    }
    if (bx > 16) {
      *smem = mySum = mySum + smem[16];
    }
    if (bx > 8) {
      *smem = mySum = mySum + smem[8];
    }
    if (bx > 4) {
      *smem = mySum = mySum + smem[4];
    }
    if (bx > 2) {
      *smem = mySum = mySum + smem[2];
    }
    if (bx > 1) {
      *smem = mySum = mySum + smem[1];
    }
  }

  if (tid == 0) buf_array_p[j * oPitch + blockIdx.x] = mySum;
}

template <int bx, typename real>
__global__ void kernel21(const int n, const int iPitch, const int oPitch,
                         const real* buf_in, real* buf_out) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y;
  const int tid = threadIdx.x;

  volatile __shared__ real sdata[bx];

  real mySum;

  if (i < n)
    mySum = buf_in[j * iPitch + i];
  else
    mySum = 0;

  sdata[tid] = mySum;
  __syncthreads();
  if (bx > 512) {
    if (tid < 512) {
      sdata[tid] = mySum = (mySum + sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (bx > 256) {
    if (tid < 256) {
      sdata[tid] = mySum = (mySum + sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (bx > 128) {
    if (tid < 128) {
      sdata[tid] = mySum = (mySum + sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (bx > 64) {
    if (tid < 64) {
      sdata[tid] = mySum = (mySum + sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < min(bx / 2, 32)) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile real* smem = sdata + tid;
    if (bx > 32) {
      *smem = mySum = mySum + smem[32];
    }
    if (bx > 16) {
      *smem = mySum = mySum + smem[16];
    }
    if (bx > 8) {
      *smem = mySum = mySum + smem[8];
    }
    if (bx > 4) {
      *smem = mySum = mySum + smem[4];
    }
    if (bx > 2) {
      *smem = mySum = mySum + smem[2];
    }
    if (bx > 1) {
      *smem = mySum = mySum + smem[1];
    }
  }

  if (tid == 0) {
    buf_out[j * oPitch + blockIdx.x] = mySum;
  }
}

template <typename real>
void prog0(const int& n, const int& m, real* wy, real* sy, const real* r,
           const real* d, int& itail, const int& iupdat, int& col, int& head,
           const real& dr, const int& iPitch0, const int& iPitch_i,
           const int& iPitch_j, real* buf_array_p, const int& iPitch_normal,
           hipStream_t st) {
  debugSync();
  kernel0<real><<<dim3(iDivUp(n, 512)), dim3(512), 0, st>>>(n, wy + itail, r,
                                                      iPitch0);
  debugSync();

  if (iupdat > m) {
    debugSync();
    kernel1<real><<<1, dim3(col, col), 0, st>>>(sy, iPitch_i, iPitch_j, col);
  }

  if (col > 1) {
    debugSync();
    int nblock0 = n;
    int mi = log2Up(nblock0);
    int nblock1 = iDivUp2(nblock0, mi);

    real* oFinal = sy + (col - 1) * iPitch_i;

    real* output = (nblock1 == 1) ? oFinal : buf_array_p;

    int op20 = (nblock1 == 1) ? iPitch_j : iPitch_normal;

    dynamicCall(kernel20, mi, real, nblock1, col - 1, st,
                (nblock0, head, m, col, iPitch0, op20, d, output, wy));

    nblock0 = nblock1;
    // Launch Ker 0
    while (nblock0 > 1) {
      nblock1 = iDivUp2(nblock0, mi);

      real* input = output;

      output = (nblock1 == 1) ? oFinal : (output + nblock0);

      int op20 = (nblock1 == 1) ? iPitch_j : iPitch_normal;
      dynamicCall(kernel21, mi, real, nblock1, col - 1, st,
                  (nblock0, iPitch_normal, op20, input, output));

      nblock0 = nblock1;
    }
    debugSync();
  }
  hipMemcpyAsync(sy + (col - 1) * iPitch0 + col - 1, &dr, sizeof(real),
                  hipMemcpyHostToDevice, st);
  debugSync();
}

#define INST_HELPER(real)                                                      \
  template void prog0<real>(const int&, const int&, real*, real*, const real*, \
                            const real*, int&, const int&, int&, int&,         \
                            const real&, const int&, const int&, const int&,   \
                            real*, const int&, hipStream_t);

INST_HELPER(double);
INST_HELPER(float);

};  // namespace matupd
};  // namespace cuda
};  // namespace lbfgsbcuda