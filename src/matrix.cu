#include "matrix.hcu"

matrix::matrix(float *A, uint m, uint n)
{
    this.m = m;
    this.n = n;
    uint sz = m * n * sizeof(float);
    hipMalloc(&dA, sz);
    hipMemcpy(dA, A, sz, hipMemcpyDeviceToHost);
}

__device__ inline float matrix::get(uint i, uint j)
{
    return dA[i * k + j];
}
