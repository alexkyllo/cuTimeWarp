
#include <hip/hip_runtime.h>
/** Take the softmin of 3 elements
 * @param a The first element
 * @param b The second element
 * @param c The third element
 * @param gamma The smoothing factor
 */
__device__ float softmin(float a, float b, float c, const float gamma)
{
    a /= -gamma;
    b /= -gamma;
    c /= -gamma;
    float max_of = max(max(a, b), c);
    float sum = exp(a - max_of) + exp(b - max_of) + exp(c - max_of);

    return -gamma * (log(sum) + max_of);
}