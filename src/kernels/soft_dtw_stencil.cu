#include "hip/hip_runtime.h"
#include "helper_functions.cuh"
//#include "soft_dtw_stencil.cuh"

/** Kernel function for computing Soft DTW on pairwise Euclidean
 * distance matrix for multivariate time series with CUDA.
 * Uses a shared memory stencil for caching the previous diagonal
 * Input D should be a __device__ array.
 * This naive version only works for sequence lengths <= 1024 i.e. can fit in
 * a single threadblock.
 * Each threadblock computes DTW for a pair of time series
 * Each thread can process one anti-diagonal.
 * @param D A 3D tensor of pairwise squared Euclidean distance matrices
 * between time series
 * @param R An m+2 x n+2 array that will be filled with the alignments
 * @param cost The total path costs will be written to this array of length nD
 * @param nD The number of distance matrices in D and its leading dimension
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__global__ void softdtw_stencil(float *D, float *R, float *cost, uint nD,
                                uint m, uint n, float gamma)
{
    // dynamic shared memory diagonal buffer array for caching the previous
    // diagonals.
    // length is (m + 2) + (m + 1) + (m) because it needs to store three
    // diagonals of R and the longest diagonal is (m+2)
    extern __shared__ float stencil[];
    const uint tx = threadIdx.x;
    const uint bx = blockIdx.x;
    uint bD = bx * m * n;
    uint bD2 = bx * (m + 2) * (n + 2);

    // block size = max(m, n) (length of longest diagonal)
    // number of antidiagonals is 2 * max(m,n) - 1
    const uint passes = 2 * blockDim.x - 1;

    // each pass is one diagonal of the distance matrix
    for (uint p = 0; p < passes; p++)
    {
        uint jj = max(0, min(p - tx, n - 1));
        uint i = tx + 1;
        uint j = jj + 1;
        // calculate the length of current diagonal that this thread is on

        // check if the thread is on the current diagonal and in-bounds
        if (tx + jj == p && (tx < m && jj < n))
        {
            // load a diagonal into shared memory
            // TODO: figure out how to index into the stencil
            // synchronize to make sure shared mem is done loading
            __syncthreads();
            float c = D[bD + (i - 1) * n + j - 1];
            // read the elements of R from the stencil
            float r1 = R[bD2 + (i - 1) * (n + 2) + j];
            float r2 = R[bD2 + i * (n + 2) + j - 1];
            float r3 = R[bD2 + (i - 1) * (n + 2) + j - 1];
            double prev_min = softmin(r1, r2, r3, gamma);
            // write the current element of R back to the stencil
            R[bD2 + i * (n + 2) + j] = c + prev_min;
        }

        // after a diagonal is no longer used, write that portion of R in
        // shared memory back to global memory
        __syncthreads();
    }
    // R[m,n] is the best path total cost, write this from the stencil
    // back to the cost array in global memory
    if (tx == 0)
    {
        cost[bx] = R[bD2 + m * (n + 2) + n];
    }
}