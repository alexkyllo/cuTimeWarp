#include "hip/hip_runtime.h"
#include "helper_functions.cuh"
#include "soft_dtw_diagonal_major.cuh"
#include <stdio.h>
void print_diag(const char *X, const uint m, const uint n)
{
    for (uint k = 0; k < m + n - 1; k++)
    {
        for (uint j = 0; j <= k; j++)
        {
            uint i = k - j;
            if (i < m && j < n)
            {
                std::cout << X[i * n + j] << " ";
            }
        }
        std::cout << "\n";
    }
}

__global__ void convert_diagonal(float *D, float *DD, uint m, uint n)
{
    const uint tx = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = tx % n;
    uint i = (tx - j) / n;
    // new i is the antidiagonal ordinal, sum of i and j
    uint dest_i = i + j;
    // new j = j if in upper left half, else j-dist from leading antidiagonal
    uint dest_j = j - max(0, (int)dest_i - (int)m + 1);
    DD[dest_i * m + dest_j] = D[i * n + j];
}

__host__ void convert_diagonal_major(float *D, float *DD, uint m, uint n)
{
    uint T = m * n;
    uint TPB = min(T, 1024);
    uint B = (T + TPB - 1) / TPB;
    convert_diagonal<<<B, TPB>>>(D, DD, m, n);
    cudaErrchk(hipDeviceSynchronize());
}

/** Kernel function for computing "naive" Soft DTW on pairwise Euclidean
 * distance matrix for multivariate time series with CUDA. Input D should be a
 * __device__ array.
 * This version assumes D is a diagonal-major array where m and n are the
 * dimensions of the original row-major array. m x n becomes (m+n-1) x min(m,n).
 * It also assumes R is a diagonal-major array where (m+2) and (n+2) are the
 * dimensions of the original row-major array.
 * (m+2) x (n+2) becomes (m+n+3) x min(m+2,n+2)
 * This naive version only works for sequence lengths <= 1024 i.e. can fit in
 * a single threadblock.
 * Assumes only a single threadblock in the kernel launch.
 * Each thread can process one anti-diagonal.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array that will be filled with the alignments
 * @param cost The total path cost will be written to this address
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__global__ void _softdtw_diagonal_kernel(float *D, float *R, float *cost,
                                         uint m, uint n, float gamma)
{
    const uint tx = threadIdx.x;
    const uint bx = blockDim.x;
    // block size = min(m, n) (length of longest diagonal)
    // number of antidiagonals is m+n-1
    const uint passes = m + n - 1;

    for (uint p = 0; p < passes; p++)
    {
        // uint jj = max(0, min(p - tx, n - 1));
        // uint dest_i = tx + jj;
        // uint dest_j = jj - max(0, (int)dest_i - (int)(m + 3));
        // uint i = dest_i + 1;
        // uint j = dest_j + 1;

        uint jj = max(0, min(p - tx, n - 1));
        uint old_i = tx + 1;
        uint old_j = jj + 1;

        uint i = old_i + old_j;
        uint j = old_j - max(0, (int)i - (int)(m + 3));

        if (tx + jj == p && (tx < m && jj < n))
        {
            float cost = D[(i - 2) * bx + j - 1];     // 1,0
            float r1 = R[(i - 1) * (bx + 2) + j];     // 1,1
            float r2 = R[(i - 2) * (bx + 2) + j - 1]; // 2, 0
            float r3 = R[(i - 1) * (bx + 2) + j - 1];
            double prev_min = softmin(r1, r2, r3, gamma);
            R[i * (bx + 2) + j] = cost + prev_min;
            if (tx == 0)
            {
                printf("pass %d tid %d reading %.2f from D[%d, %d]\n", p, tx,
                       cost, i - 2, j - 1);
                printf("pass %d tid %d reading %.2f from R[%d, %d]\n", p, tx,
                       r1, i - 1, j);
                printf("pass %d tid %d reading %.2f from R[%d, %d]\n", p, tx,
                       r2, i - 2, j - 1);
                printf("pass %d tid %d reading %.2f from R[%d, %d]\n", p, tx,
                       r3, i - 1, j - 1);
                printf(
                    "pass %d tx %d jj %d i %d j %d writing %.2f to R[%d, %d]\n",
                    p, tx, jj, i, j, cost + prev_min, i, j);
            }
        }
        __syncthreads();
    }
    if (tx == 0)
    {
        *cost = R[m * (bx + 2) + n];
    }
}

__global__ void softdtw_diagonal_kernel(float *D, float *R, float *cost, uint m,
                                        uint n, float gamma)
{
    const uint tx = threadIdx.x;
    const uint bx = blockDim.x;
    // block size = min(m, n) (length of longest diagonal)
    // number of antidiagonals is m+n-1
    // D is now (m+n-1) x min(m,n)
    // R is now (m+n+3) x min(m+1,n+1)
    const uint passes = m + n - 1;

    for (uint p = 0; p < passes; p++)
    {
        uint ii = max(0, (int)p - (int)tx);
        uint past_mid = max(0, (int)p - (int)bx + 1);
        uint i = ii + 1 - past_mid;
        uint j = tx + 1 + past_mid;

        if (tx + ii <= p && j <= n)
        {
            // convert i,j to diagonal-major coordinates
            // new j = j if in upper left half, else j-dist from leading
            // antidiagonal
            uint di = (i - 1) + (j - 1);
            uint dj = j - 1 - past_mid;
            uint ri = i + j;
            uint rj = j - past_mid;
            uint r1j = rj - 1;
            uint r2j = rj - 1;
            uint r3j = rj;

            // If we are past the antidiagonal, need to increment the previous
            // cell references
            if (p >= bx)
            {
                r1j++;
                r2j++;
                r3j++;
            }
            if (p > bx)
            {
                r1j++;
            }

            float cost = D[di * bx + dj];
            float r1 = R[di * (bx + 2) + r1j];
            float r2 = R[(ri - 1) * (bx + 2) + r2j];
            float r3 = R[(ri - 1) * (bx + 2) + r3j];
            double prev_min = softmin(r1, r2, r3, gamma);
            R[ri * (bx + 2) + rj] = cost + prev_min;
        }
        __syncthreads();
        if (tx == 0)
        {
            *cost = R[(m + n) * (bx + 2) + 1];
        }
    }
}