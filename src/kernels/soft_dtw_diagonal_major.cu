#include "hip/hip_runtime.h"
#include "helper_functions.cuh"
#include "soft_dtw_diagonal_major.cuh"
#include <stdio.h>
void print_diag(const char *X, const uint m, const uint n)
{
    for (uint k = 0; k < m + n - 1; k++)
    {
        for (uint j = 0; j <= k; j++)
        {
            uint i = k - j;
            if (i < m && j < n)
            {
                // std::cout << X[i * n + j] << " ";
            }
        }
        // std::cout << "\n";
    }
}

__global__ void convert_diagonal(float *D, float *DD, uint m, uint n)
{
    const uint tx = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = tx % n;
    uint i = (tx - j) / n;
    uint dest_i = i + j;
    uint dest_j = j - max(0, (int)dest_i - (int)m + 1);
    DD[dest_i * m + dest_j] = D[i * n + j];
}

#define cudaErrchk(ans)                                                        \
    {                                                                          \
        GPUAssert((ans), __FILE__, __LINE__);                                  \
    }
inline void GPUAssert(hipError_t code, const char *file, int line,
                      bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
        {
            exit(code);
        }
    }
}

__host__ void convert_diagonal_major(float *D, float *DD, uint m, uint n)
{
    uint T = m * n;
    uint TPB = min(T, 1024);
    uint B = (T + TPB - 1) / TPB;
    convert_diagonal<<<B, TPB>>>(D, DD, m, n);
    cudaErrchk(hipDeviceSynchronize());
}

/** Kernel function for computing "naive" Soft DTW on pairwise Euclidean
 * distance matrix for multivariate time series with CUDA. Input D should be a
 * __device__ array.
 * This naive version only works for sequence lengths <= 1024 i.e. can fit in
 * a single threadblock.
 * Assumes only a single threadblock in the kernel launch.
 * Each thread can process one anti-diagonal.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array that will be filled with the alignments
 * @param cost The total path cost will be written to this address
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__global__ void softdtw_diagonal_kernel(float *D, float *R, float *cost, uint m,
                                        uint n, float gamma)
{
    const uint tx = threadIdx.x;
    // block size = max(m, n) (length of longest diagonal)
    // number of antidiagonals is 2 * max(m,n) - 1
    const uint passes = 2 * blockDim.x - 2;

    for (uint p = 0; p < passes; p++)
    {
        uint jj = max(0, min(p - tx, n - 1));
        uint i = tx + 1;
        uint j = jj + 1;

        if (tx + jj == p && (tx < m && jj < n))
        {
            float cost = D[(i - 1) * n + j - 1];
            float r1 = R[(i - 1) * (n + 2) + j];
            float r2 = R[i * (n + 2) + j - 1];
            float r3 = R[(i - 1) * (n + 2) + j - 1];
            double prev_min = softmin(r1, r2, r3, gamma);
            R[i * (n + 2) + j] = cost + prev_min;
        }
        __syncthreads();
    }
    if (tx == 0)
    {
        *cost = R[m * (n + 2) + n];
    }
}
