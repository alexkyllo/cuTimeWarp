#include "hip/hip_runtime.h"
/** CUDA implementation of Soft DTW
 *  @file soft_dtw.cu
 */
#include "kernels/euclid_dist.cuh"
#include "kernels/helper_functions.cuh"
#include "kernels/soft_dtw_naive.cuh"
#include "kernels/soft_dtw_naive_multi.cuh"
#include <cmath>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>

typedef unsigned int uint;

/** Host function for computing Soft DTW on pairwise Euclidean distance matrix
 * for multivariate time series with CUDA.
 * Input D should be a __device__ array.
 * Only a single block is used. m and n must each be no longer than 1024.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 device array that will be filled with alignment values.
 * @param nD The number of distance matrices in D and its leading dimension
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__host__ float softdtw_cuda_naive(float *D, float *R, uint m, uint n,
                                  float gamma)
{
    size_t m2n2 = (m + 2) * (n + 2);
    // Launch a kernel to fill matrix R with infinity
    const int inf_tpb = 256;
    int inf_blocks = (m2n2 + inf_tpb - 1) / m2n2;
    fill_matrix_inf<<<inf_blocks, inf_tpb>>>(
        R, m + 2, n + 2, std::numeric_limits<float>::infinity());

    dim3 B = dim3(1);
    dim3 TPB = dim3(max(m, n));
    float path_cost;
    float *d_path_cost;
    hipMalloc(&d_path_cost, sizeof(float));
    // Launch the kernel
    softdtw_naive_kernel<<<B, TPB>>>(D, R, d_path_cost, m, n, gamma);
    // Copy the path cost back to host
    hipMemcpy(&path_cost, d_path_cost, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_path_cost);

    return path_cost;
}

/** Host function for computing Soft DTW on pairwise Euclidean distance matrix
 * for multivariate time series with CUDA.
 * Input D should be a __device__ array of dimension (nD x m x n).
 * Each threadblock computes DTW for a pair of time series
 * m and n must each be no longer than 1024.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An (nD x (m+2) x (n+2)) device array to fill with alignment values.
 * @param costs A length nD array that will be filled with the pairwise costs
 * @param nD The number of distance matrices in D and its leading dimension
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__host__ void softdtw_cuda_naive_multi(float *D, float *R, float *costs,
                                       uint nD, uint m, uint n, float gamma)
{
    size_t m2n2 = nD * (m + 2) * (n + 2);
    // Launch a kernel to fill matrix R with infinity
    const int inf_tpb = 256;
    int inf_blocks = (m2n2 + inf_tpb - 1) / m2n2;
    fill_matrix_inf<<<inf_blocks, inf_tpb>>>(
        R, (m + 2) * (n + 2), nD, std::numeric_limits<float>::infinity());

    dim3 B = dim3(nD);
    dim3 TPB = dim3(max(m, n));
    float *d_path_cost;
    hipMalloc(&d_path_cost, nD * sizeof(float));
    // Launch the kernel
    softdtw_naive_kernel_multi<<<B, TPB>>>(D, R, d_path_cost, nD, m, n, gamma);
    // Copy the path cost back to host
    hipMemcpy(costs, d_path_cost, nD * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_path_cost);
}

/** Host function for computing SoftDTW gradient by backpropagation
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array of alignment values.
 * @param E An m x n array that will be filled with the gradient values.
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__host__ void softdtw_grad_cuda_naive(float *D, float *R, float *E, uint m,
                                      uint n, float gamma)
{
    // Allocate larger temporary device arrays for D and E
    float *D_;
    hipMalloc(&D_, (m + 1) * (n + 1) * sizeof(float));
    hipMemset(D_, 0, (m + 1) * (n + 1) * sizeof(float));
    // Copy each row of D to D_
    for (uint i = 0; i < m; i++)
    {
        hipMemcpy(&D_[i * (n + 1)], &D[i * n], n * sizeof(float),
                   hipMemcpyDeviceToDevice);
    }
    float *E_;
    hipMalloc(&E_, (m + 2) * (n + 2) * sizeof(float));
    hipMemset(E_, 0, (m + 2) * (n + 2) * sizeof(float));

    // D_ is m+1 x n+1
    // R and E_ are m+2 x n+2
    // fill the last row and column of D with 0
    // fill the last row and column of R with -inf
    float neg_inf = -INFINITY;
    for (uint i = 1; i < m + 1; i++)
    {
        hipMemset(&D_[(i - 1) * (n + 1) + n], 0, sizeof(float));
        hipMemcpy(&R[i * (n + 2) + n + 1], &neg_inf, sizeof(float),
                   hipMemcpyHostToDevice);
    }

    for (uint j = 1; j < n + 1; j++)
    {
        hipMemset(&D_[m * (n + 1) + (j - 1)], 0, sizeof(float));
        hipMemcpy(&R[(m + 1) * (n + 2) + j], &neg_inf, sizeof(float),
                   hipMemcpyHostToDevice);
    }

    // Set last element of E to 1
    float one = 1.0;
    hipMemcpy(&E_[(m + 1) * (n + 2) + n + 1], &one, sizeof(float),
               hipMemcpyHostToDevice);

    hipMemcpy(&R[(m + 1) * (n + 2) + n + 1], &R[m * (n + 2) + n],
               sizeof(float), hipMemcpyDeviceToDevice);

    // Set last element of D to 0
    hipMemset(&D[m * (n + 1) + n], 0, sizeof(float));

    dim3 B = dim3(1);
    dim3 TPB = dim3(max(m, n));
    softdtw_grad_naive_kernel<<<B, TPB>>>(D_, R, E_, m, n, gamma);

    // Copy E_ back to E without the first and last row and column
    for (uint i = 0; i < m; i++)
    {
        hipMemcpy(&E[i * n], &E_[(i + 1) * (n + 2) + 1], n * sizeof(float),
                   hipMemcpyDeviceToDevice);
    }
    hipFree(D_);
    hipFree(E_);
}

// TODO: Barycenter computation (average time series under SoftDTW geometry)
// through gradient descent with SoftDTW as loss function
// TODO: 1-nearest neighbor classification function
