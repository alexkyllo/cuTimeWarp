#include "hip/hip_runtime.h"
/** CUDA implementation of Soft DTW
 *  @file soft_dtw.cu
 */
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

typedef unsigned int uint;

#define cudaErrchk(ans)                                                        \
    {                                                                          \
        GPUAssert((ans), __FILE__, __LINE__);                                  \
    }
inline void GPUAssert(hipError_t code, const char *file, int line,
                      bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
        {
            exit(code);
        }
    }
}

/** Take the softmin of 3 elements
 * @param a The first element
 * @param b The second element
 * @param c The third element
 * @param gamma The smoothing factor
 */
__device__ float softmin(float a, float b, float c, const float gamma)
{
    a /= -gamma;
    b /= -gamma;
    c /= -gamma;
    float max_of = max(max(a, b), c);
    float sum = exp(a - max_of) + exp(b - max_of) + exp(c - max_of);

    return -gamma * (log(sum) + max_of);
}

/** A wrapper for hipblasSgemm that works on row-major matrices by transposing
 *  A, B and C should be __device__ arrays
 *  @param A Input matrix of dimensions m * k
 *  @param B Input matrix of dimensions k * n
 *  @param C Result matrix of dimensions m * n
 *  @param m Height of matrix A and matrix C
 *  @param k Width of matrix A and height of matrix B
 *  @param n Width of matrix B and matrix C
 *  @param alpha A scalar to elementwise multiply A by
 */
__host__ void sgemm_cublas(const float *A, const float *B, float *C,
                           const uint m, const uint k, const uint n,
                           const float alpha)
{
    const float beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // call cuBLAS to multiply transposed matrices B^T * A
    // (input is row-major but cublas expects column major)
    hipblasSgemm(handle,      // cublas handle
                HIPBLAS_OP_T, // transpose first matrix
                HIPBLAS_OP_N, // tranpose second matrix
                n,           // rows in first matrix
                m,           // columns in second matrix
                k,           // columns in first matrix
                &alpha,      // scalar for first matrix
                B,           // first matrix
                k,           // stride of first matrix
                A,           // second matrix
                k,           // stride of second matrix
                &beta,       // scalar for C
                C,           // result matrix
                n            // stride of result matrix
    );
    hipblasDestroy(handle);
}

/** CUDA kernel to compute the squared euclidean norm of matrix X
 *  @param m Height (rows) of matrix X
 *  @param k Width (columns) of matrix X
 *  @param XX a length m vector for the result
 */
__global__ void sq_euclid_norm(const uint m, const uint k, const float *X,
                               float *XX)
{
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m)
    {
        for (uint j = 0; j < k; j++)
        {
            float x = X[i * k + j];
            XX[i] += x * x;
        }
    }
}

/** CUDA kernel to compute the euclidean distance between two sets of vectors
 *  X and Y by using the euclidian norms, i.e. X*X + Y*Y - 2X*Y
 *  @param m The length of vectors in X
 *  @param n The length of vectors in Y
 *  @param XX Squared Euclidean norm of X
 *  @param YY Squared Euclidean norm of Y
 *  @param XY 2 * X * Y^T (matrix multiplication result)
 *  @param D The result euclidean distance matrix with dimensions (m x n)
 */
__global__ void euclid_dist(const uint m, const uint n, const float *XX,
                            const float *YY, const float *XY, float *D)
{
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m)
    {
        for (uint j = 0; j < n; j++)
        {
            D[i * n + j] = XX[i] + YY[j] - (XY[i * n + j]);
        }
    }
}

/** Host function to compute the Squared Euclidean distance between two sets of
 vectors
 *  X and Y by using the euclidian norms, i.e. X*X + Y*Y - 2X*Y
 *  Inputs X, Y, D should be device vectors
 *  @param X A set of vectors of length (row count) m
 *  @param Y A set of vectors of length (row count) n
 *  @param m The length of vectors in X
 *  @param n The length of vectors in Y

 *  @param YY Squared Euclidean norm of Y
 *  @param XY 2 * X * Y^T (matrix multiplication result)
 *  @param D The result euclidean distance matrix with dimensions (m x n)
 */
__host__ void sq_euclid_dist(const float *X, const float *Y, float *D,
                             const uint m, const uint n, const uint k)
{
    // TODO: change this to work on device arrays only
    float *dX;
    float *dY;
    float *dD;
    float *XX; // = new float[m]{0};
    float *YY; // = new float[n]{0};
    float *XY; // = new float[m * n]{0};
    size_t size_m = m * sizeof(float);
    size_t size_n = n * sizeof(float);
    size_t size_mn = n * size_m;
    size_t size_mk = k * size_m;
    size_t size_nk = k * size_n;
    hipMalloc(&dD, size_mn);
    hipMalloc(&dX, size_mk);
    hipMalloc(&dY, size_nk);
    hipMalloc(&XX, size_m);
    hipMalloc(&YY, size_n);
    hipMalloc(&XY, size_mn);
    hipMemset(XX, 0, size_m);
    hipMemset(YY, 0, size_n);
    hipMemset(XY, 0, size_mn);
    hipMemset(dD, 0, size_mn);
    hipMemcpy(dX, X, size_mk, hipMemcpyHostToDevice);
    hipMemcpy(dY, Y, size_nk, hipMemcpyHostToDevice);

    uint block_size = min(m, 1024);
    uint grid_size = (m + block_size - 1) / block_size;
    // compute squared euclidean norm of X
    sq_euclid_norm<<<grid_size, block_size>>>(m, k, dX, XX);
    block_size = min(n, 1024);
    grid_size = (n + block_size - 1) / block_size;
    sq_euclid_norm<<<block_size, grid_size>>>(n, k, dY, YY);

    // // compute (2*X)*YT
    sgemm_cublas(dX, dY, XY, m, k, n, 2.0);

    block_size = min(m, 1024);
    grid_size = (m + block_size - 1) / block_size;
    euclid_dist<<<block_size, grid_size>>>(m, n, XX, YY, XY, dD);
    cudaErrchk(hipMemcpy(D, dD, size_mn, hipMemcpyDeviceToHost));
    hipFree(dD);
    hipFree(XX);
    hipFree(YY);
    hipFree(XY);
}
