#include "hip/hip_runtime.h"
/** CUDA implementation of Soft DTW
 *  @file soft_dtw.cu
 */
#include <cmath>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>

typedef unsigned int uint;

/** Take the softmin of 3 elements
 * @param a The first element
 * @param b The second element
 * @param c The third element
 * @param gamma The smoothing factor
 */
__device__ float softmin(float a, float b, float c, const float gamma)
{
    a /= -gamma;
    b /= -gamma;
    c /= -gamma;
    float max_of = max(max(a, b), c);
    float sum = exp(a - max_of) + exp(b - max_of) + exp(c - max_of);

    return -gamma * (log(sum) + max_of);
}

/** A wrapper for hipblasSgemm that works on row-major matrices by transposing
 *  A, B and C should be __device__ arrays
 *  @param A Input matrix of dimensions m * k
 *  @param B Input matrix of dimensions k * n
 *  @param C Result matrix of dimensions m * n
 *  @param m Height of matrix A and matrix C
 *  @param k Width of matrix A and height of matrix B
 *  @param n Width of matrix B and matrix C
 *  @param alpha A scalar to elementwise multiply A by
 */
__host__ void sgemm_cublas(const float *A, const float *B, float *C,
                           const uint m, const uint k, const uint n,
                           const float alpha)
{
    const float beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // call cuBLAS to multiply transposed matrices B^T * A
    // (input is row-major but cublas expects column major)
    hipblasSgemm(handle,      // cublas handle
                HIPBLAS_OP_T, // transpose first matrix
                HIPBLAS_OP_N, // tranpose second matrix
                n,           // rows in first matrix
                m,           // columns in second matrix
                k,           // columns in first matrix
                &alpha,      // scalar for first matrix
                B,           // first matrix
                k,           // stride of first matrix
                A,           // second matrix
                k,           // stride of second matrix
                &beta,       // scalar for C
                C,           // result matrix
                n            // stride of result matrix
    );
    hipblasDestroy(handle);
}

/** CUDA kernel to compute the squared euclidean norm of matrix X
 *  @param m Height (rows) of matrix X
 *  @param k Width (columns) of matrix X
 *  @param XX a length m vector for the result
 */
__global__ void sq_euclid_norm(const uint m, const uint k, const float *X,
                               float *XX)
{
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m)
    {
        for (uint j = 0; j < k; j++)
        {
            float x = X[i * k + j];
            XX[i] += x * x;
        }
    }
}

/** CUDA kernel to compute the euclidean distance between two Euclidean norm
 * vectors XX and YY, i.e. X*X + Y*Y - 2X*Y
 *  @param m The length of vectors in X
 *  @param n The length of vectors in Y
 *  @param XX Squared Euclidean norm of X
 *  @param YY Squared Euclidean norm of Y
 *  @param XY 2 * X * Y^T (matrix multiplication result)
 *  @param D The result euclidean distance matrix with dimensions (m x n)
 */
__global__ void euclid_dist(const uint m, const uint n, const float *XX,
                            const float *YY, const float *XY, float *D)
{
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < m)
    {
        for (uint j = 0; j < n; j++)
        {
            D[i * n + j] = XX[i] + YY[j] - (XY[i * n + j]);
        }
    }
}

/** Host function to compute the Squared Euclidean distance between two sets of
 *  column vectors (e.g. two multivariate time series)
 *  X and Y by using the euclidian norms, i.e. X*X + Y*Y - 2X*Y
 *  Inputs X, Y, D should be __device__ arrays.
 *  @param X A set of vectors of length (row count) m
 *  @param Y A set of vectors of length (row count) n
 *  @param D A result array for the distance matrix of dimension (m x n)
 *  @param m The length of vectors in X
 *  @param n The length of vectors in Y
 *  @param k The number of vectors in X and Y (columns)
 */
__host__ void sq_euclid_dist(const float *X, const float *Y, float *D,
                             const uint m, const uint n, const uint k)
{
    float *XX;
    float *YY;
    float *XY;
    size_t size_m = m * sizeof(float);
    size_t size_n = n * sizeof(float);
    size_t size_mn = n * size_m;
    hipMalloc(&XX, size_m);
    hipMalloc(&YY, size_n);
    hipMalloc(&XY, size_mn);
    hipMemset(XX, 0, size_m);
    hipMemset(YY, 0, size_n);
    hipMemset(XY, 0, size_mn);
    hipMemset(D, 0, size_mn);

    uint block_size = min(m, 1024);
    uint grid_size = (m + block_size - 1) / block_size;
    // compute squared euclidean norm of X
    sq_euclid_norm<<<grid_size, block_size>>>(m, k, X, XX);
    block_size = min(n, 1024);
    grid_size = (n + block_size - 1) / block_size;
    sq_euclid_norm<<<block_size, grid_size>>>(n, k, Y, YY);

    // compute (2*X)*YT
    sgemm_cublas(X, Y, XY, m, k, n, 2.0);

    block_size = min(m, 1024);
    grid_size = (m + block_size - 1) / block_size;
    euclid_dist<<<block_size, grid_size>>>(m, n, XX, YY, XY, D);
    hipFree(XX);
    hipFree(YY);
    hipFree(XY);
}

/** Host function to compute all pairwise squared Euclidean distances between
 *  two sets of time series so that we can compute Soft-DTW
 *  on many distance matrices in parallel.
 *  Inputs X, Y, D should be __device__ arrays.
 *  @param X A set of nX vectors of length (row count) m x k (column count)
 *  @param Y A set of nY vectors of length (row count) n x k (column count)
 *  @param D A result array for the distance matrix of dimension (m x n)
 *  @param nX The number of time series in batch X
 *  @param nY The number of time series in batch Y
 *  @param m The length of vectors in X
 *  @param n The length of vectors in Y
 *  @param k The number of vectors in X and Y (columns)
 */
__host__ void sq_euclid_dist_multi(const float *X, const float *Y, float *D,
                                   const uint nX, const uint nY, const uint m,
                                   const uint n, const uint k)
{
    // TODO work in progress, needs testing, probably going to be slow
    // Maybe rather than computing this for all pairs and then softdtw,
    // it would be faster to compute parallelize across the pairs once and
    // compute the distance matrix and softdtw for each pair independently?
    float *XX; // nX x m
    float *YY; // nY x n
    float *XY; // (nX x nY) x m x n
    size_t size_mx = nX * m * sizeof(float);
    size_t size_ny = nY * n * sizeof(float);
    size_t size_mnxy = nX * m * size_ny;
    hipMalloc(&XX, size_mx);
    hipMalloc(&YY, size_ny);
    hipMalloc(&XY, size_mnxy);
    hipMemset(XX, 0, size_mx);
    hipMemset(YY, 0, size_ny);
    hipMemset(XY, 0, size_mnxy);
    hipMemset(D, 0, size_mnxy);

    uint block_size_m = min(m, 1024);
    uint grid_size_m = (m + block_size_m - 1) / block_size_m;
    uint block_size_n = min(n, 1024);
    uint grid_size_n = (n + block_size_n - 1) / block_size_n;
    // compute squared euclidean norm of X
    // is a loop the best way to do this or can we write one kernel to compute
    // multiple norms in parallel?
    // Need to use hipStreamCreate to run kernels in the loop in parallel?
    for (uint i = 0; i < m; i++)
    {
        sq_euclid_norm<<<grid_size_m, block_size_m>>>(m, k, &X[i * (m * k)],
                                                      &XX[i * m]);
    }
    for (uint i = 0; i < n; i++)
    {
        sq_euclid_norm<<<block_size_n, grid_size_n>>>(n, k, &Y[i * (n * k)],
                                                      &YY[i * n]);
    }
    hipDeviceSynchronize();
    const float beta = 0.0;
    const float alpha = 2.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // Compute 2*X*Y^T for each X and Y
    for (uint i = 0; i < nX; i++)
    {
        for (uint j = 0; j < nY; j++)
        {
            // call cuBLAS to multiply transposed matrices B^T * A
            // (input is row-major but cublas expects column major)
            hipblasSgemm(handle,                    // cublas handle
                        HIPBLAS_OP_T,               // transpose first matrix
                        HIPBLAS_OP_N,               // tranpose second matrix
                        n,                         // rows in first matrix
                        m,                         // columns in second matrix
                        k,                         // columns in first matrix
                        &alpha,                    // scalar for first matrix
                        &Y[j * (n * k)],           // first matrix
                        k,                         // stride of first matrix
                        &X[i * (m * k)],           // second matrix
                        k,                         // stride of second matrix
                        &beta,                     // scalar for C
                        &XY[(i * nX + j) * m * n], // result matrix
                        n                          // stride of result matrix
            );
            // compute XX + YY - 2XY for each pair of X and Y
            euclid_dist<<<block_size_m, grid_size_m>>>(
                m, n, &XX[i * m], &YY[j * n], &XY[(i * nX + j) * m * n],
                &D[(i * nX + j) * m * n]);
        }
    }
    hipblasDestroy(handle);
    hipFree(XX);
    hipFree(YY);
    hipFree(XY);
}

// TODO: Compute squared euclidean distance for many time series in parallel
// TODO: Write a kernel that can handle an additional dimension of D (distance
// matrix).

/** Host function for retrieving the number of SMs on the GPU device
 *  Useful for limiting the # of threadblocks to the # of SMs in a kernel launch
 *  @param device_num The device number, default 0
 *  @return the SM count
 */
__host__ uint get_device_sm_count(uint device_num = 0)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_num);
    return deviceProp.multiProcessorCount;
}

/** Kernel function for computing "naive" Soft DTW on pairwise Euclidean
 * distance matrix for multivariate time series with CUDA. Input D should be a
 * __device__ array.
 * This naive version only works for sequence lengths <= 1024 i.e. can fit in
 * a single threadblock.
 * Assumes only a single threadblock in the kernel launch.
 * Each thread can process one anti-diagonal.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array that will be filled with the alignments
 * @param cost The total path cost will be written to this address
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__global__ void softdtw_naive_kernel(float *D, float *R, float *cost, uint m,
                                     uint n, float gamma)
{
    const uint tx = threadIdx.x;
    // block size = max(m, n) (length of longest diagonal)
    // number of antidiagonals is 2 * max(m,n) - 1
    const uint passes = 2 * blockDim.x - 1;

    for (uint p = 0; p < passes; p++)
    {
        uint jj = max(0, min(p - tx, n - 1));
        uint i = tx + 1;
        uint j = jj + 1;

        if (tx + jj == p && (tx < m && jj < n))
        {
            float cost = D[(i - 1) * n + j - 1];
            float r1 = R[(i - 1) * (n + 2) + j];
            float r2 = R[i * (n + 2) + j - 1];
            float r3 = R[(i - 1) * (n + 2) + j - 1];
            double prev_min = softmin(r1, r2, r3, gamma);
            R[i * (n + 2) + j] = cost + prev_min;
        }
        __syncthreads();
    }
    if (tx == 0)
    {
        *cost = R[m * (n + 2) + n];
    }
}

/** Kernel function for computing "naive" Soft DTW on pairwise Euclidean
 * distance matrix for multivariate time series with CUDA.
 * Input D should be a __device__ array.
 * This naive version only works for sequence lengths <= 1024 i.e. can fit in
 * a single threadblock.
 * Each threadblock computes DTW for a pair of time series
 * Each thread can process one anti-diagonal.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array that will be filled with the alignments
 * @param cost The total path costs will be written to this array of length nD
 * @param nD The number of distance matrices in D and its leading dimension
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__global__ void softdtw_naive_kernel_multi(float *D, float *R, float *cost,
                                           uint nD, uint m, uint n, float gamma)
{
    const uint tx = threadIdx.x;
    const uint bx = blockIdx.x;
    uint bD = bx * m * n;
    uint bD2 = bx * (m + 2) * (n + 2);

    // block size = max(m, n) (length of longest diagonal)
    // number of antidiagonals is 2 * max(m,n) - 1
    const uint passes = 2 * blockDim.x - 1;

    for (uint p = 0; p < passes; p++)
    {
        uint jj = max(0, min(p - tx, n - 1));
        uint i = tx + 1;
        uint j = jj + 1;

        if (tx + jj == p && (tx < m && jj < n))
        {
            float c = D[bD + (i - 1) * n + j - 1];
            float r1 = R[bD2 + (i - 1) * (n + 2) + j];
            float r2 = R[bD2 + i * (n + 2) + j - 1];
            float r3 = R[bD2 + (i - 1) * (n + 2) + j - 1];
            double prev_min = softmin(r1, r2, r3, gamma);
            R[bD2 + i * (n + 2) + j] = c + prev_min;
        }
        __syncthreads();
    }
    if (tx == 0)
    {
        cost[bx] = R[bD2 + m * (n + 2) + n];
    }
}

/** Kernel function for computing "naive" SoftDTW gradient by backpropagation
 * This naive version only works for sequence lengths <= 1024 i.e. can fit in
 * a single threadblock.
 * Assumes only a single threadblock in the kernel launch.
 * Each thread can process one anti-diagonal.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array that will be filled with the alignment
 * values.
 * @param E An m+2 x n+2 array that will be filled with the gradient values.
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__global__ void softdtw_grad_naive_kernel(float *D, float *R, float *E, uint m,
                                          uint n, float gamma)
{
    const uint tx = threadIdx.x;
    const uint passes = 2 * blockDim.x - 1;

    for (uint p = 0; p < passes; p++)
    {
        uint backward_p = passes - p - 1;
        uint jj = max(0, min(backward_p - tx, n - 1));
        uint i = tx + 1;
        uint j = jj + 1;
        if (tx + jj == backward_p && (tx < m && jj < n))
        {
            if (isinf(R[i * (n + 2) + j]))
            {
                R[i * (n + 2) + j] = -INFINITY;
            }
            float r0 = R[i * (n + 2) + j];
            float a =
                exp((R[(i + 1) * (n + 2) + j] - r0 - D[i * (n + 1) + (j - 1)]) /
                    gamma);
            float b =
                exp((R[i * (n + 2) + (j + 1)] - r0 - D[(i - 1) * (n + 1) + j]) /
                    gamma);
            float c =
                exp((R[(i + 1) * (n + 2) + j + 1] - r0 - D[i * (n + 1) + j]) /
                    gamma);
            E[i * (n + 2) + j] = E[(i + 1) * (n + 2) + j] * a +
                                 E[i * (n + 2) + j + 1] * b +
                                 E[(i + 1) * (n + 2) + j + 1] * c;
        }
        __syncthreads();
    }
}

/** Kernel function for computing tiled Soft DTW on pairwise Euclidean distance
 * matrix for multivariate time series with CUDA. Input D should be a
 * __device__ array.
 * This naive version only works for sequence length <= 1024.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array that will be filled with the alignments
 * @param cost The total path cost will be written to this address
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__global__ void softdtw_tiled_kernel(float *D, float *R, float *cost, uint m,
                                     uint n, float gamma)
{
    // TODO
    // Divide R into tiles
    // Each tile depends on the tiles to its top, left, and top-left
    // Assign one thread to spin on the signal variable for this tile
    // Process the tile diagonally from upper left to lower right
    // using a loop counter to keep track of fully processed diagonals
    // and while loop and syncthreads to spin on it
    // Write to the signal variables to signal the next tiles
}

/** Kernel to fill a matrix with infinity except for index 0 = 0.0
 *  to initialize the DTW cost matrix
 */
__global__ void fill_matrix_inf(float *A, int width, int height, float val)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = idx; i < width * height; i += gridDim.x * blockDim.x)
    {
        A[i] = val;
        if (i % width == 0)
            A[i] = 0.0;
    }
}

/** Host function for computing Soft DTW on pairwise Euclidean distance matrix
 * for multivariate time series with CUDA.
 * Input D should be a __device__ array.
 * Only a single block is used. m and n must each be no longer than 1024.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 device array that will be filled with alignment values.
 * @param nD The number of distance matrices in D and its leading dimension
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__host__ float softdtw_cuda_naive(float *D, float *R, uint m, uint n,
                                  float gamma)
{
    size_t m2n2 = (m + 2) * (n + 2);
    // Launch a kernel to fill matrix R with infinity
    const int inf_tpb = 256;
    int inf_blocks = (m2n2 + inf_tpb - 1) / m2n2;
    fill_matrix_inf<<<inf_blocks, inf_tpb>>>(
        R, m + 2, n + 2, std::numeric_limits<float>::infinity());

    dim3 B = dim3(1);
    dim3 TPB = dim3(max(m, n));
    float path_cost;
    float *d_path_cost;
    hipMalloc(&d_path_cost, sizeof(float));
    // Launch the kernel
    softdtw_naive_kernel<<<B, TPB>>>(D, R, d_path_cost, m, n, gamma);
    // Copy the path cost back to host
    hipMemcpy(&path_cost, d_path_cost, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_path_cost);

    return path_cost;
}

/** Host function for computing Soft DTW on pairwise Euclidean distance matrix
 * for multivariate time series with CUDA.
 * Input D should be a __device__ array of dimension (nD x m x n).
 * Each threadblock computes DTW for a pair of time series
 * m and n must each be no longer than 1024.
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An (nD x (m+2) x (n+2)) device array to fill with alignment values.
 * @param costs A length nD array that will be filled with the pairwise costs
 * @param nD The number of distance matrices in D and its leading dimension
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__host__ void softdtw_cuda_naive_multi(float *D, float *R, float *costs,
                                       uint nD, uint m, uint n, float gamma)
{
    size_t m2n2 = nD * (m + 2) * (n + 2);
    // Launch a kernel to fill matrix R with infinity
    const int inf_tpb = 256;
    int inf_blocks = (m2n2 + inf_tpb - 1) / m2n2;
    fill_matrix_inf<<<inf_blocks, inf_tpb>>>(
        R, (m + 2) * (n + 2), nD, std::numeric_limits<float>::infinity());

    dim3 B = dim3(nD);
    dim3 TPB = dim3(max(m, n));
    float *d_path_cost;
    hipMalloc(&d_path_cost, nD * sizeof(float));
    // Launch the kernel
    softdtw_naive_kernel_multi<<<B, TPB>>>(D, R, d_path_cost, nD, m, n, gamma);
    // Copy the path cost back to host
    hipMemcpy(costs, d_path_cost, nD * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_path_cost);
}

/** Host function for computing SoftDTW gradient by backpropagation
 * @param D The pairwise squared Euclidean distance array of two time series
 * @param R An m+2 x n+2 array of alignment values.
 * @param E An m x n array that will be filled with the gradient values.
 * @param m Length of first time series
 * @param n Length of second time series
 * @param gamma SoftDTW smoothing parameter
 */
__host__ void softdtw_grad_cuda_naive(float *D, float *R, float *E, uint m,
                                      uint n, float gamma)
{
    // Allocate larger temporary device arrays for D and E
    float *D_;
    hipMalloc(&D_, (m + 1) * (n + 1) * sizeof(float));
    hipMemset(D_, 0, (m + 1) * (n + 1) * sizeof(float));
    // Copy each row of D to D_
    for (uint i = 0; i < m; i++)
    {
        hipMemcpy(&D_[i * (n + 1)], &D[i * n], n * sizeof(float),
                   hipMemcpyDeviceToDevice);
    }
    float *E_;
    hipMalloc(&E_, (m + 2) * (n + 2) * sizeof(float));
    hipMemset(E_, 0, (m + 2) * (n + 2) * sizeof(float));

    // D_ is m+1 x n+1
    // R and E_ are m+2 x n+2
    // fill the last row and column of D with 0
    // fill the last row and column of R with -inf
    float neg_inf = -INFINITY;
    for (uint i = 1; i < m + 1; i++)
    {
        hipMemset(&D_[(i - 1) * (n + 1) + n], 0, sizeof(float));
        hipMemcpy(&R[i * (n + 2) + n + 1], &neg_inf, sizeof(float),
                   hipMemcpyHostToDevice);
    }

    for (uint j = 1; j < n + 1; j++)
    {
        hipMemset(&D_[m * (n + 1) + (j - 1)], 0, sizeof(float));
        hipMemcpy(&R[(m + 1) * (n + 2) + j], &neg_inf, sizeof(float),
                   hipMemcpyHostToDevice);
    }

    // Set last element of E to 1
    float one = 1.0;
    hipMemcpy(&E_[(m + 1) * (n + 2) + n + 1], &one, sizeof(float),
               hipMemcpyHostToDevice);

    hipMemcpy(&R[(m + 1) * (n + 2) + n + 1], &R[m * (n + 2) + n],
               sizeof(float), hipMemcpyDeviceToDevice);

    // Set last element of D to 0
    hipMemset(&D[m * (n + 1) + n], 0, sizeof(float));

    dim3 B = dim3(1);
    dim3 TPB = dim3(max(m, n));
    softdtw_grad_naive_kernel<<<B, TPB>>>(D_, R, E_, m, n, gamma);

    // Copy E_ back to E without the first and last row and column
    for (uint i = 0; i < m; i++)
    {
        hipMemcpy(&E[i * n], &E_[(i + 1) * (n + 2) + 1], n * sizeof(float),
                   hipMemcpyDeviceToDevice);
    }
    hipFree(D_);
    hipFree(E_);
}

// TODO: Barycenter computation (average time series under SoftDTW geometry)
// through gradient descent with SoftDTW as loss function

// TODO: 1-nearest neighbor classification function
